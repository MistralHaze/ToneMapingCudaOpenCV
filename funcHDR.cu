#include "hip/hip_runtime.h"
#include <hip/device_functions.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>

#include <iostream>
#include <iomanip>

#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 32

const bool using_thrust = false;

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

template <typename T>
void check(T err, const char* const func, const char* const file,
	const int line) {
	if (err != hipSuccess) {
		std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		exit(1);
	}
}

//From CUDA EXAMPLE LIBRARY
// simple routine to print contents of a vector
template <typename Vector>
void print_vector(const std::string& name, const Vector& v)
{
	typedef typename Vector::value_type T;
	std::cout << "  " << std::setw(20) << name << "  ";
	thrust::copy(v.begin(), v.end(), std::ostream_iterator<T>(std::cout, " "));
	std::cout << std::endl;
}

struct calculateBin
{
	const float numBins;
	const float minLum;
	const float lumRange;
	int* d_histogram;

	calculateBin(float _numBins, float _minLum, float _lumRange, int* _histogram) : numBins(_numBins), minLum(_minLum), lumRange(_lumRange), d_histogram(_histogram) {}

	__device__ float operator()(const float& logLuminance)  
	{
		int bin = roundf((logLuminance - minLum) / lumRange * numBins);
		atomicAdd(&(d_histogram[bin]), 1);
		return 1;
	}
};

void findMinMaxLuminancy(const float* const d_logLuminance,
	float& min_logLum,
	float& max_logLum,
	float& luminanceRange,
	const size_t numRows,
	const size_t numCols,
	const dim3& blockSize,
	const dim3& gridSize);


void cdf_with_thrust(const float* const d_logLuminance,
	unsigned int* const d_cdf,
	float& min_logLum,
	float& max_logLum,
	const size_t numRows,
	const size_t numCols,
	const size_t numBins);

__global__ void get_min_max(const float* const d_logLuminance, float* d_minLogLumPerBlock, float* const d_maxLogLumPerBlock, const size_t numRows, const size_t numCols)
{
	__shared__ float sharedMax[BLOCK_WIDTH * BLOCK_HEIGHT];
	__shared__ float sharedMin[BLOCK_WIDTH * BLOCK_HEIGHT];

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	// 0 - total threads en ejecucion
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	// 0 - num threads por bloque
	const int shared_pos = threadIdx.y * blockDim.x + threadIdx.x;

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
	{
		sharedMax[shared_pos] = 0.0f;
		sharedMin[shared_pos] = INFINITY;
		return;
	}
	else
	{
		sharedMax[shared_pos] = d_logLuminance[thread_1D_pos];
		sharedMin[shared_pos] = sharedMax[shared_pos];
	}

	__syncthreads();

	for (unsigned int s = (blockDim.x * blockDim.y) >> 1; s > 0; s >>= 1)
	{
		if (shared_pos < s)
		{
			//Max
			if (sharedMax[shared_pos] < sharedMax[shared_pos + s])
			{
				sharedMax[shared_pos] = sharedMax[shared_pos + s];
			}

			//Min
			if (sharedMin[shared_pos] > sharedMin[shared_pos + s])
			{
				sharedMin[shared_pos] = sharedMin[shared_pos + s];
			}
		}
		__syncthreads();
	}

	if (shared_pos == 0)
	{
		const unsigned int blockId = blockIdx.y * ceil((float)numCols / BLOCK_WIDTH) + blockIdx.x;

		d_maxLogLumPerBlock[blockId] = sharedMax[0];
		d_minLogLumPerBlock[blockId] = sharedMin[0];
	}
}

/*
* This function is intended to work with the output of get_min_max.
* Since that function outputs max values of each block in an array, we have to find
* the max value from there. This kernel helps if the array is too big to search it
* efficiently in CPU.
*/
__global__ void get_min_max_aux(float* const d_minLogLumPerBlock, float* const d_maxLogLumPerBlock, const size_t size)
{
	__shared__ float sharedMax[BLOCK_WIDTH * BLOCK_HEIGHT];
	__shared__ float sharedMin[BLOCK_WIDTH * BLOCK_HEIGHT];

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	// 0 - total threads en ejecucion
	const int thread_1D_pos = thread_2D_pos.x * blockDim.y + thread_2D_pos.y;

	// 0 - num threads por bloque
	const int shared_pos = threadIdx.y * blockDim.x + threadIdx.x;

	if (thread_1D_pos >= size)
	{
		sharedMax[shared_pos] = 0.0f;
		sharedMin[shared_pos] = INFINITY;
		return;
	}
	else
	{
		sharedMax[shared_pos] = d_maxLogLumPerBlock[thread_1D_pos];
		sharedMin[shared_pos] = d_minLogLumPerBlock[thread_1D_pos];
	}

	__syncthreads();

	for (unsigned int s = (blockDim.x * blockDim.y) >> 1; s > 0; s >>= 1)
	{
		if (shared_pos < s)
		{
			//Max
			if (sharedMax[shared_pos] < sharedMax[shared_pos + s])
			{
				sharedMax[shared_pos] = sharedMax[shared_pos + s];
			}

			//Min
			if (sharedMin[shared_pos] > sharedMin[shared_pos + s])
			{
				sharedMin[shared_pos] = sharedMin[shared_pos + s];
			}
		}
		__syncthreads();
	}

	if (shared_pos == 0)
	{
		const unsigned int blockId = blockIdx.y * ceil((float)size / BLOCK_WIDTH) + blockIdx.x;

		d_maxLogLumPerBlock[blockId] = sharedMax[0];
		d_minLogLumPerBlock[blockId] = sharedMin[0];
	}
}

__global__ void generateHistogram(const float* const d_logLuminance, const size_t numBins, unsigned int* const d_histogram,
	const float luminanceRange, const float min_logLum, const size_t numRows, const size_t numCols)
{
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	// 0 - total threads en ejecucion
	//const int thread_1D_pos = thread_2D_pos.x * numCols + thread_2D_pos.y;
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	//All threads handle blockDim.x* gridDim.x  consecutive elements
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows) return;

	//bin = (Lum[i] - lumMin) / lumRange * numBins;
	//numBins value is 1024 so we need to settle the range 0 - 1023
	unsigned int bin = roundf((d_logLuminance[thread_1D_pos] - min_logLum) / luminanceRange * (numBins - 1));
	atomicAdd(&(d_histogram[bin]), 1);
}

__global__ void exclusive_scan(unsigned int* const d_histogram, unsigned int* const d_cdf, const size_t numBins, const size_t numRows, const size_t numCols)
{
	__shared__ int shared_cdf[BLOCK_HEIGHT * BLOCK_WIDTH];

	// 0 - num threads por bloque
	const int shared_pos = threadIdx.y * blockDim.x + threadIdx.x;

	// Assign shared memory
	shared_cdf[shared_pos] = d_histogram[shared_pos];

	// Reduction
	unsigned int offset = 1;
	for (unsigned int s = numBins >> 1; s > 0; s >>= 1)
	{
		__syncthreads();
		if (shared_pos < s)
		{
			int ai = offset * (2 * shared_pos + 1) - 1;
			int bi = offset * (2 * shared_pos + 2) - 1;
			shared_cdf[bi] += shared_cdf[ai];
		}

		offset <<= 1;
	}

	if (shared_pos == 0)
	{
		shared_cdf[numBins - 1] = 0;
	}

	// Reverse
	for (int s = 1; s < numBins; s <<= 1)
	{
		offset >>= 1;

		__syncthreads();
		if (shared_pos < s)
		{
			int ai = offset * (2 * shared_pos + 1) - 1;
			int bi = offset * (2 * shared_pos + 2) - 1;
			int temp = shared_cdf[ai];
			shared_cdf[ai] = shared_cdf[bi];
			shared_cdf[bi] += temp;
		}
	}

	__syncthreads();

	d_cdf[shared_pos] = shared_cdf[shared_pos];

}

void calculate_cdf(const float* const d_logLuminance,
	unsigned int* const d_cdf,
	float& min_logLum,
	float& max_logLum,
	const size_t numRows,
	const size_t numCols,
	const size_t numBins)
{
	if (using_thrust)
	{
		cdf_with_thrust(d_logLuminance, d_cdf, min_logLum, max_logLum, numRows, numCols, numBins);
	}
	else
	{
		float luminanceRange;

		// Estableciendo memoria para el histograma
		unsigned int* d_histogram;
		checkCudaErrors(hipMalloc(&d_histogram, sizeof(unsigned int) * numBins));
		checkCudaErrors(hipMemset(d_histogram, 0, sizeof(unsigned int) * numBins));

		// Tama�o de bloque y de grid
		const dim3 blockSize(BLOCK_WIDTH, BLOCK_HEIGHT, 1);
		const dim3 gridSize(ceil((float)numCols / BLOCK_WIDTH), ceil((float)numRows / BLOCK_HEIGHT), 1);
		// TODO
		//1) Encontrar el valor m�ximo y m�nimo de luminancia en min_logLum and max_logLum a partir del canal logLuminance
		//Crear valor intermedio para trabajar
		//2) Obtener el rango a representar
		findMinMaxLuminancy(d_logLuminance, min_logLum, max_logLum, luminanceRange, numRows, numCols, blockSize, gridSize);

		//3) Generar un histograma de todos los valores del canal logLuminance usando la formula
		generateHistogram << <gridSize, blockSize >> > (d_logLuminance, numBins, d_histogram, luminanceRange, min_logLum, numRows, numCols);

		//4) Realizar un exclusive scan en el histograma para obtener la distribuci�n acumulada (cdf)
		//de los valores de luminancia. Se debe almacenar en el puntero c_cdf
		exclusive_scan << <dim3(1, 1, 1), blockSize >> > (d_histogram, d_cdf, numBins, numRows, numCols);

	}
}

void cdf_with_thrust(const float* const d_logLuminance,
	unsigned int* const d_cdf,
	float& min_logLum,
	float& max_logLum,
	const size_t numRows,
	const size_t numCols,
	const size_t numBins)
{
	//Create a pointer that thrust can work with
	thrust::device_ptr<const float> d_logLuminance_ptr(d_logLuminance);

	//Declare the thrust vector
	thrust::device_vector<float> thrust_logLuminance(d_logLuminance_ptr, d_logLuminance_ptr + numCols * numRows);

	//Min and max
	thrust::device_ptr<const float>  max = thrust::max_element(thrust::device, d_logLuminance_ptr, d_logLuminance_ptr + numCols * numRows);
	thrust::device_ptr<const float>  min = thrust::min_element(thrust::device, d_logLuminance_ptr, d_logLuminance_ptr + numCols * numRows);

	float thrust_lumRange = *max - *min;
	min_logLum = *min;
	max_logLum = *max;

	if (thrust_lumRange == 0)
	{
		std::cerr << "Computed range of luminance was 0. Returning" << std::endl;
		return;
	}

	//Declare the thrust vector
	thrust::device_vector<int> thrust_histogram(numBins);
	int* thrust_histogram_ptr = thrust::raw_pointer_cast(&thrust_histogram[0]);
	thrust::fill(thrust_histogram.begin(), thrust_histogram.end(), 0);

	calculateBin op(numBins, min_logLum, thrust_lumRange , thrust_histogram_ptr);

	//Ahora thrustLogLuminance tiene las posiciones donde se debe aumentar en uno en el array
	thrust::transform(thrust_logLuminance.begin(), thrust_logLuminance.end(), thrust_logLuminance.begin(),	op);

	thrust::exclusive_scan(thrust_histogram.begin(), thrust_histogram.end(), d_cdf); // in-place scan
}

void findMinMaxLuminancy(const float* const d_logLuminance,
	float& min_logLum,
	float& max_logLum,
	float& luminanceRange,
	const size_t numRows,
	const size_t numCols,
	const dim3& blockSize,
	const dim3& gridSize)
{
	float* d_maxLogLumPerBlock, * d_minLogLumPerBlock;
	float* d_maxLogLum, * d_minLogLum;

	checkCudaErrors(hipMalloc(&d_maxLogLumPerBlock, sizeof(float) * gridSize.x * gridSize.y));
	checkCudaErrors(hipMalloc(&d_minLogLumPerBlock, sizeof(float) * gridSize.x * gridSize.y));

	checkCudaErrors(hipMalloc(&d_maxLogLum, sizeof(float)));
	checkCudaErrors(hipMalloc(&d_minLogLum, sizeof(float)));

	get_min_max << < gridSize, blockSize >> > (d_logLuminance, d_minLogLumPerBlock, d_maxLogLumPerBlock, numRows, numCols);

	//Obtener max y min a partir de los max min de cada bloque
	int threadsPerBlock = BLOCK_WIDTH * BLOCK_HEIGHT;
	int numThreads = gridSize.x * gridSize.y;
	while (numThreads > 1)
	{
		int numBlocks = ceil((float)numThreads / threadsPerBlock);
		dim3 gridSizeAux(numBlocks, 1, 1);

		get_min_max_aux << < gridSizeAux, blockSize >> > (d_minLogLumPerBlock, d_maxLogLumPerBlock, numThreads);

		numThreads = numBlocks;
	}

	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpy((void*)&max_logLum, (void*)&d_maxLogLumPerBlock[0], sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((void*)&min_logLum, (void*)&d_minLogLumPerBlock[0], sizeof(float), hipMemcpyDeviceToHost));

	// Set Luminance Range
	luminanceRange = max_logLum - min_logLum;
	if (luminanceRange == 0)
	{
		std::cerr << "Computed range of luminance was 0. Returning" << std::endl;
		return;
	}

	//std::cout << " After Kernel max " << max_logLum << std::endl;
	//std::cout << " After Kernel min " << min_logLum << std::endl;

	checkCudaErrors(hipFree(d_maxLogLumPerBlock));
	checkCudaErrors(hipFree(d_minLogLumPerBlock));
	checkCudaErrors(hipFree(d_maxLogLum));
	checkCudaErrors(hipFree(d_minLogLum));
}